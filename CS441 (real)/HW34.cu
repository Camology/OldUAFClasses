
#include <hip/hip_runtime.h>
#include "stdio.h"
#define COLUMNS 8 
#define ROWS 8

__global__ void add(int *a, int *b) {
	int cacheIndex = threadIdx.x;
	int i = blockDim.x/2;
	while (i > 0) {
		if (cacheIndex < i) {
			a[blockIdx.x * COLUMNS +cacheIndex] += a[blockIdx.x * COLUMNS + cacheIndex +i];
		}
		__syncthreads();
		i /=2; 
	}
	if (cacheIndex == 0)
		b[blockIdx.x] = a[blockIdx.x * COLUMNS];
		
}

int main() {
	int a[ROWS][COLUMNS], b[COLUMNS];
	int *dev_a;
	int *dev_b;
	int sum =0;
	int cudSum =0;
	hipMalloc((void **)&dev_a, ROWS*COLUMNS*sizeof(int));
	hipMalloc((void **)&dev_b, COLUMNS*sizeof(int));
	
	for(int y=0; y<ROWS; y++) {
		for(int x=0; x<COLUMNS; x++){
			a[y][x] = x+y;
			sum+= x+y;
		}
	}
	printf("Sum is: %d \n", sum);
	hipMemcpy(dev_a, a, ROWS*COLUMNS*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, COLUMNS*sizeof(int), hipMemcpyHostToDevice);
	
	dim3 thread(COLUMNS,ROWS);
	add<<<8,8>>>(dev_a,dev_b);
	hipMemcpy(b,dev_b, COLUMNS*sizeof(int),hipMemcpyDeviceToHost);
	for (int i=0; i<ROWS; i++) {
		cudSum+= b[i];
	} 
	printf("cuda sum is: %d \n", cudSum);

	hipFree(dev_a);
	hipFree(dev_b);
	return 0;

}
