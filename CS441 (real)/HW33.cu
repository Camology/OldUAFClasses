
#include <hip/hip_runtime.h>
#include "stdio.h"
#define COLUMNS 4 
#define ROWS 3

__global__ void add(int *a, int *b) {
	int x = threadIdx.x;
	int sum =0;
	for (unsigned int i=0; i < ROWS; i++){
		sum += a[i*COLUMNS + x];
	}
	b[x] = sum;
		
}

int main() {
	int a[ROWS][COLUMNS], b[COLUMNS];
	int *dev_a;
	int *dev_b;
	int sum =0;
	int cudSum =0;
	hipMalloc((void **)&dev_a, ROWS*COLUMNS*sizeof(int));
	hipMalloc((void **)&dev_b, COLUMNS*sizeof(int));
	
	for(int y=0; y<ROWS; y++) {
		for(int x=0; x<COLUMNS; x++){
			a[y][x] = x;
			sum+= x;
		}
	}
	printf("Sum is: %d \n", sum);
	hipMemcpy(dev_a, a, ROWS*COLUMNS*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, COLUMNS*sizeof(int), hipMemcpyHostToDevice);
	
	dim3 thread(COLUMNS,ROWS);
	add<<<1,COLUMNS>>>(dev_a,dev_b);
	hipMemcpy(b,dev_b, COLUMNS*sizeof(int),hipMemcpyDeviceToHost);
	for (int i=0; i<COLUMNS; i++) {
		cudSum+= b[i];
	} 
	printf("cuda sum is: %d \n", cudSum);
	hipFree(dev_a);
	hipFree(dev_b);
	return 0;

}
